#include "hip/hip_runtime.h"
#include "core.h"
#include "parameters.h"

#include <cmath>
#include <cstdlib> // rand()

#include <hip/hip_runtime.h>
#include "forces_gpu.h"
#include "hip/hip_runtime_api.h"


void init_pos(float* rx, float* ry, float* rz, const float rho)
{
    // inicialización de las posiciones de los átomos en un cristal FCC

    float a = cbrt(4.0 / rho); // cbrt=cube root
    int nucells = ceil(cbrt((float)N / 4.0));
    int idx = 0;

    for (int i = 0; i < nucells; i++) {
        for (int j = 0; j < nucells; j++) {
            for (int k = 0; k < nucells; k++) {
                rx[idx] = i * a; // x
                ry[idx] = j * a; // y
                rz[idx] = k * a; // z
                    // del mismo átomo
                rx[idx + 1] = (i + 0.5) * a;
                ry[idx + 1] = (j + 0.5) * a;
                rz[idx + 1] = k * a;

                rx[idx + 2] = (i + 0.5) * a;
                ry[idx + 2] = j * a;
                rz[idx + 2] = (k + 0.5) * a;

                rx[idx + 3] = i * a;
                ry[idx + 3] = (j + 0.5) * a;
                rz[idx + 3] = (k + 0.5) * a;

                idx += 4;
            }
        }
    }
}


void init_vel(float* vx, float* vy, float* vz, float* temp, float* ekin)
{
    // inicialización de velocidades aleatorias

    float sf, sumvx = 0.0, sumvy = 0.0, sumvz = 0.0, sumv2 = 0.0;
    for (int i = 0; i < N; i++) {
        vx[i] = rand() / (float)RAND_MAX - 0.5;
        vy[i] = rand() / (float)RAND_MAX - 0.5;
        vz[i] = rand() / (float)RAND_MAX - 0.5;
        sumvx += vx[i];
        sumvy += vy[i];
        sumvz += vz[i];
        sumv2 += vx[i] * vx[i] + vy[i] * vy[i] + vz[i] * vz[i];
    }

    sumvx /= (float)N;
    sumvy /= (float)N;
    sumvz /= (float)N;
    *temp = sumv2 / (3.0 * N);
    *ekin = 0.5 * sumv2;
    sf = sqrt(T0 / *temp);

    for (int i = 0; i < N; i++) { // elimina la velocidad del centro de masa
        // y ajusta la temperatura
        vx[i] = (vx[i] - sumvx) * sf;
        vy[i] = (vy[i] - sumvy) * sf;
        vz[i] = (vz[i] - sumvz) * sf;
    }
}

static float pbc(float cordi, const float cell_length)
{
    // condiciones periodicas de contorno coordenadas entre [0,L)
    if (cordi <= 0) {
        cordi += cell_length;
    } else if (cordi > cell_length) {
        cordi -= cell_length;
    }
    return cordi;
}


void velocity_verlet(float* rx, float* ry, float* rz, float* vx,
                     float* vy, float* vz, float* fx, float* fy,
                     float* fz, float* epot, float* ekin, float* pres,
                     float* temp, const float rho, const float V,
                     const float L)
{

    for (int i = 0; i < N; i++) { // actualizo posiciones
        rx[i] += vx[i] * DT + 0.5 * fx[i] * DT * DT;
        ry[i] += vy[i] * DT + 0.5 * fy[i] * DT * DT;
        rz[i] += vz[i] * DT + 0.5 * fz[i] * DT * DT;

        rx[i] = pbc(rx[i], L);
        ry[i] = pbc(ry[i], L);
        rz[i] = pbc(rz[i], L);

        vx[i] += 0.5 * fx[i] * DT;
        vy[i] += 0.5 * fy[i] * DT;
        vz[i] += 0.5 * fz[i] * DT;
    }

    for (int j = 0; j < N; j++) {
        fx[j] = 0.0;
        fy[j] = 0.0;
        fz[j] = 0.0;
    }
    *epot = 0;
    *pres = *temp * rho;
    {
            float *epot_aux;
            float *pres_aux;
            float *ptr_Temp;


            checkCudaError(hipMallocManaged(&epot_aux, sizeof(float *)));
            checkCudaError(hipMallocManaged(&pres_aux, sizeof(float *)));
            checkCudaError(hipMallocManaged(&ptr_Temp, sizeof(float *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = *temp;

        // for (int i = 0; i < N - 1; i += 1) {
            launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, rho, V, L); // actualizo fuerzas
        // }
        *epot += *epot_aux;
        *pres += *pres_aux;

        checkCudaError(hipFree(epot_aux));
        checkCudaError(hipFree(pres_aux));
        checkCudaError(hipFree(ptr_Temp));
    }


    float sumv2 = 0.0;
    for (int i = 0; i < N; i++) { // actualizo velocidades
        vx[i] += 0.5 * fx[i] * DT;
        vy[i] += 0.5 * fy[i] * DT;
        vz[i] += 0.5 * fz[i] * DT;

        sumv2 += vx[i] * vx[i] + vy[i] * vy[i] + vz[i] * vz[i];
    }
    *ekin = 0.5 * sumv2;
    *temp = sumv2 / (3.0 * N);
}
