#include "hip/hip_runtime.h"
#include "core.h"
#include "parameters.h"

#include <cmath>
#include <cstdlib> // rand()

#include <hip/hip_runtime.h>
#include "forces_gpu.h"
#include "hip/hip_runtime_api.h"


void init_pos(double* rx, double* ry, double* rz, const double rho)
{
    // inicialización de las posiciones de los átomos en un cristal FCC

    double a = cbrt(4.0 / rho); // cbrt=cube root
    int nucells = ceil(cbrt((double)N / 4.0));
    int idx = 0;

    for (int i = 0; i < nucells; i++) {
        for (int j = 0; j < nucells; j++) {
            for (int k = 0; k < nucells; k++) {
                rx[idx] = i * a; // x
                ry[idx] = j * a; // y
                rz[idx] = k * a; // z
                    // del mismo átomo
                rx[idx + 1] = (i + 0.5) * a;
                ry[idx + 1] = (j + 0.5) * a;
                rz[idx + 1] = k * a;

                rx[idx + 2] = (i + 0.5) * a;
                ry[idx + 2] = j * a;
                rz[idx + 2] = (k + 0.5) * a;

                rx[idx + 3] = i * a;
                ry[idx + 3] = (j + 0.5) * a;
                rz[idx + 3] = (k + 0.5) * a;

                idx += 4;
            }
        }
    }
}


void init_vel(double* vx, double* vy, double* vz, double* temp, double* ekin)
{
    // inicialización de velocidades aleatorias

    double sf, sumvx = 0.0, sumvy = 0.0, sumvz = 0.0, sumv2 = 0.0;
    for (int i = 0; i < N; i++) {
        vx[i] = rand() / (double)RAND_MAX - 0.5;
        vy[i] = rand() / (double)RAND_MAX - 0.5;
        vz[i] = rand() / (double)RAND_MAX - 0.5;
        sumvx += vx[i];
        sumvy += vy[i];
        sumvz += vz[i];
        sumv2 += vx[i] * vx[i] + vy[i] * vy[i] + vz[i] * vz[i];
    }

    sumvx /= (double)N;
    sumvy /= (double)N;
    sumvz /= (double)N;
    *temp = sumv2 / (3.0 * N);
    *ekin = 0.5 * sumv2;
    sf = sqrt(T0 / *temp);

    for (int i = 0; i < N; i++) { // elimina la velocidad del centro de masa
        // y ajusta la temperatura
        vx[i] = (vx[i] - sumvx) * sf;
        vy[i] = (vy[i] - sumvy) * sf;
        vz[i] = (vz[i] - sumvz) * sf;
    }
}

static double pbc(double cordi, const double cell_length)
{
    // condiciones periodicas de contorno coordenadas entre [0,L)
    if (cordi <= 0) {
        cordi += cell_length;
    } else if (cordi > cell_length) {
        cordi -= cell_length;
    }
    return cordi;
}


void velocity_verlet(double* rx, double* ry, double* rz, double* vx,
                     double* vy, double* vz, double* fx, double* fy,
                     double* fz, double* epot, double* ekin, double* pres,
                     double* temp, const double rho, const double V,
                     const double L)
{

    for (int i = 0; i < N; i++) { // actualizo posiciones
        rx[i] += vx[i] * DT + 0.5 * fx[i] * DT * DT;
        ry[i] += vy[i] * DT + 0.5 * fy[i] * DT * DT;
        rz[i] += vz[i] * DT + 0.5 * fz[i] * DT * DT;

        rx[i] = pbc(rx[i], L);
        ry[i] = pbc(ry[i], L);
        rz[i] = pbc(rz[i], L);

        vx[i] += 0.5 * fx[i] * DT;
        vy[i] += 0.5 * fy[i] * DT;
        vz[i] += 0.5 * fz[i] * DT;
    }

    for (int j = 0; j < N; j++) {
        fx[j] = 0.0;
        fy[j] = 0.0;
        fz[j] = 0.0;
    }
    *epot = 0;
    *pres = *temp * rho;
    {
            double *epot_aux;
            double *pres_aux;
            double *ptr_Temp;


            checkCudaCall(hipMallocManaged(&epot_aux, sizeof(double *)));
            checkCudaCall(hipMallocManaged(&pres_aux, sizeof(double *)));
            checkCudaCall(hipMallocManaged(&ptr_Temp, sizeof(double *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = *temp;

        // for (int i = 0; i < N - 1; i += 1) {
            launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, rho, V, L); // actualizo fuerzas
        // }
        *epot += *epot_aux;
        *pres += *pres_aux;

        checkCudaCall(hipFree(epot_aux));
        checkCudaCall(hipFree(pres_aux));
        checkCudaCall(hipFree(ptr_Temp));
    }


    double sumv2 = 0.0;
    for (int i = 0; i < N; i++) { // actualizo velocidades
        vx[i] += 0.5 * fx[i] * DT;
        vy[i] += 0.5 * fy[i] * DT;
        vz[i] += 0.5 * fz[i] * DT;

        sumv2 += vx[i] * vx[i] + vy[i] * vy[i] + vz[i] * vz[i];
    }
    *ekin = 0.5 * sumv2;
    *temp = sumv2 / (3.0 * N);
}
