#include "hip/hip_runtime.h"
// Variables necesarias

// #include <hip/hip_runtime.h> // no se porque esto rompe todo

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" // checkCudaError
#include "parameters.h"
#include <cstdio>
// #include <hip/hip_runtime.h>

// #include <hipcub/hipcub.hpp>

# define CUDA_WARP_SIZE 32


__device__ void minimum_image(float cordi, const float cell_length, float* result)
{
    // imagen más cercana
    if (cordi <= -0.5 * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5 * cell_length) {
        cordi -= cell_length;
    }

    *result = cordi;
}

__global__ void forces(const float* rx,
                       const float* ry,
                       const float* rz,
                       float* fx,
                       float* fy,
                       float* fz,
                       float* epot,
                       float* pres,
                       const float* temp,
                       const float rho,
                       const float V,
                       const float L
                       )
{

    float rcut2 = RCUT * RCUT;
    const float RCUT12 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;

    const float RCUT6 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;
    const float ECUT = 4.0 * (1 / (RCUT12)-1 / (RCUT6));
    // //#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

    float fxi = 0.0;
    float fyi = 0.0;
    float fzi = 0.0;
    float epot_partial = 0.0;
    float pres_vir_partial = 0.0;


    unsigned int j =  threadIdx.x;
    unsigned int row =  blockIdx.x;
    for(; j < N ;j+= CUDA_WARP_SIZE){

        if (j != row) {
            float xi = rx[row];
            float yi = ry[row];
            float zi = rz[row];

            float xj = rx[j];
            float yj = ry[j];
            float zj = rz[j];

            float rxd = xi - xj;
            float ryd = yi - yj;
            float rzd = zi - zj;

            minimum_image(rxd, L, &rxd);
            minimum_image(ryd, L, &ryd);
            minimum_image(rzd, L, &rzd);

            float rij2 = rxd * rxd + ryd * ryd + rzd * rzd;

            if (rij2 <= rcut2) {
                float r2inv = 1.0 / rij2;
                float r6inv = r2inv * r2inv * r2inv;

                float fr = 24.0 * r2inv * r6inv * (2.0 * r6inv - 1.0);

                fxi += fr * rxd;
                fyi += fr * ryd;
                fzi += fr * rzd;


                epot_partial += 4.0 * r6inv * (r6inv - 1.0) - ECUT;
                pres_vir_partial += fr * rij2;
            }
        }
    }

    // fx[row]+=fxi;
    // fy[row]+=fyi;
    // fz[row]+=fzi;
    atomicAdd(&fx[row], fxi);
    atomicAdd(&fy[row], fyi);
    atomicAdd(&fz[row], fzi);

    atomicAdd(epot, epot_partial / 2);
    atomicAdd(pres, pres_vir_partial / 2 / (V * 3.0));

}


int div_ceil(int a, int b)
{
    return (a + b - 1) / b;
}


void launch_forces(const float* rx, const float* ry, const float* rz,
                   float* fx, float* fy, float* fz, float* epot,
                   float* pres, const float* temp, const float rho,
                   const float V, const float L)
{

    dim3 block(CUDA_WARP_SIZE);

    dim3 grid(N);


    forces <<<grid, block>>> (rx, ry, rz, fx, fy, fz, epot, pres, temp, rho,
                              V, L);

    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
}

