#include "hip/hip_runtime.h"
// Variables necesarias

// #include <hip/hip_runtime.h> // no se porque esto rompe todo

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" // checkCudaCall
#include "parameters.h"
#include <cstdio>
// #include <hip/hip_runtime.h>

// #include <hipcub/hipcub.hpp>



//#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

__device__ void minimum_image(double cordi, const double cell_length, double* result)
{
    // imagen más cercana
    if (cordi <= -0.5 * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5 * cell_length) {
        cordi -= cell_length;
    }

    *result = cordi;
}

// Esta función esta sacada de la wiki de Cuda. capaz sirve.


// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
// old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
//     } while (assumed != old);
//     return __longlong_as_double(old);
// }



// Algo asi capaz si se podria implementar para la reduccion con doubles.


// __global__ void sum_shared_atomic(const int *in, int n, int *out)
// {
//     __shared__ int partial_sum;

//     uint i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (threadIdx.x == 0) {
//         partial_sum = 0;
//     }

//     __syncthreads();

//     if (i < n) {
//         atomicAdd(&partial_sum, in[i]);
//     }

//     __syncthreads();

//     if (threadIdx.x == 0) {
//         atomicAdd(out, partial_sum);
//     }
// }



__global__ void forces(const double* rx,
                       const double* ry,
                       const double* rz,
                       double* fx,
                       double* fy,
                       double* fz,
                       double* epot,
                       double* pres,
                       const double* temp,
                       const double rho,
                       const double V,
                       const double L,
                       const int row)
{

    //        fx[row] = 0.0d;
    //        fy[row] = 0.0d;
    //        fz[row] = 0.0d;

    //    *epot = 0.0;
    if (threadIdx.x == 0 ){
    // printf("Soy el hilo %i", threadIdx.x); // esto funciona, solo el hilo 0 ejecuta esto
    double rcut2 = RCUT * RCUT;
    const double RCUT12 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;

    const double RCUT6 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;
    const double ECUT = 4.0 * (1 / (RCUT12)-1 / (RCUT6));
    // //#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

    double fxi = 0.0;
    double fyi = 0.0;
    double fzi = 0.0;
    double epot_partial = 0.0;
    double pres_vir_partial = 0.0;

    for (int j = 0; j < (N - 1); j++) {
        if (j != row) {
            double xi = rx[row];
            double yi = ry[row];
            double zi = rz[row];

            double xj = rx[j];
            double yj = ry[j];
            double zj = rz[j];

            double rxd = xi - xj;
            double ryd = yi - yj;
            double rzd = zi - zj;

            minimum_image(rxd, L, &rxd);
            minimum_image(ryd, L, &ryd);
            minimum_image(rzd, L, &rzd);

            double rij2 = rxd * rxd + ryd * ryd + rzd * rzd;

            if (rij2 <= rcut2) {
                double r2inv = 1.0 / rij2;
                double r6inv = r2inv * r2inv * r2inv;

                double fr = 24.0 * r2inv * r6inv * (2.0 * r6inv - 1.0);

                fxi += fr * rxd;
                fyi += fr * ryd;
                fzi += fr * rzd;


                epot_partial += 4.0 * r6inv * (r6inv - 1.0) - ECUT;
                pres_vir_partial += fr * rij2;
            }
        }
    }

    // Estas funciones no parecen andar con doubles. Se podria investigar un poco más.
    // atomicAdd(fx[row], fxi);
    // atomicAdd(fy[row], fyi);
    // atomicAdd(fz[row], fzi);

    // atomicAdd(epot, epot_partial / 2);
    // atomicAdd(pres, pres_vir_partial / 2 / (V * 3.0));

    fx[row] += fxi;

    fy[row] += fyi;
    fz[row] += fzi;
    *epot += epot_partial / 2;
    *pres += pres_vir_partial / 2 / (V * 3.0);

}
}
int div_ceil(int a, int b) {
    return (a + b - 1) / b;
}

void launch_forces(const double* rx, const double* ry, const double* rz,
                   double* fx, double* fy, double* fz, double* epot,
                   double* pres, const double* temp, const double rho,
                   const double V, const double L)
{

    // Todavia no entiendo que numero de bloques y de grilla nos conviene usar para el problema

    // int block_size = N;
    // int num_blocks = N;

    dim3 block(16);
    dim3 grid(div_ceil(N, block.x));


    // Este for probablemente no tendria que ir, deberiamos lanzar un kernel que haga esto segun el hilo en el que esta parado
    for(size_t i = 0; i < N-1; i++ ) {
    forces <<<grid, block>>> (rx, ry, rz, fx, fy, fz, epot, pres, temp, rho,
                              V, L, i);

    }
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
}

