#include "hip/hip_runtime.h"
// Variables necesarias

// #include <hip/hip_runtime.h> // no se porque esto rompe todo

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" // checkCudaCall
#include "parameters.h"
#include <cstdio>
#include <hip/hip_runtime.h>

// #include <hipcub/hipcub.hpp>



//#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

__device__ void minimum_image(double cordi, const double cell_length, double* result)
{
    // imagen más cercana
    if (cordi <= -0.5 * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5 * cell_length) {
        cordi -= cell_length;
    }

    *result = cordi;
}


__global__ void forces(const double* rx,
                       const double* ry,
                       const double* rz,
                       double* fx,
                       double* fy,
                       double* fz,
                       double* epot,
                       double* pres,
                       const double* temp,
                       const double rho,
                       const double V,
                       const double L,
                       const int row)
{

    //        fx[row] = 0.0d;
    //        fy[row] = 0.0d;
    //        fz[row] = 0.0d;

    //    *epot = 0.0;
    double rcut2 = RCUT * RCUT;
    const double RCUT12 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;

    const double RCUT6 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;
    const double ECUT = 4.0 * (1 / (RCUT12)-1 / (RCUT6));
    // //#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

    double fxi = 0.0;
    double fyi = 0.0;
    double fzi = 0.0;
    double epot_partial = 0.0;
    double pres_vir_partial = 0.0;

    for (int j = 0; j < (N - 1); j++) {
        if (j != row) {
            double xi = rx[row];
            double yi = ry[row];
            double zi = rz[row];

            double xj = rx[j];
            double yj = ry[j];
            double zj = rz[j];

            double rxd = xi - xj;
            double ryd = yi - yj;
            double rzd = zi - zj;

            minimum_image(rxd, L, &rxd);
            minimum_image(ryd, L, &ryd);
            minimum_image(rzd, L, &rzd);

            double rij2 = rxd * rxd + ryd * ryd + rzd * rzd;

            if (rij2 <= rcut2) {
                double r2inv = 1.0 / rij2;
                double r6inv = r2inv * r2inv * r2inv;

                double fr = 24.0 * r2inv * r6inv * (2.0 * r6inv - 1.0);

                fxi += fr * rxd;
                fyi += fr * ryd;
                fzi += fr * rzd;


                epot_partial += 4.0 * r6inv * (r6inv - 1.0) - ECUT;
                pres_vir_partial += fr * rij2;
            }
        }
    }


    atomicAdd(&fx[row], fxi);
    atomicAdd(&fy[row], fyi);
    atomicAdd(&fz[row], fzi);

    atomicAdd(&epot, epot_partial / 2);
    atomicAdd(&pres, pres_vir_partial / 2 / (V * 3.0));

    // fx[row] += fxi;

    // fy[row] += fyi;
    // fz[row] += fzi;
    *epot += epot_partial / 2;
    *pres += pres_vir_partial / 2 / (V * 3.0);

}



void launch_forces(const double* rx, const double* ry, const double* rz,
                   double* fx, double* fy, double* fz, double* epot,
                   double* pres, const double* temp, const double rho,
                   const double V, const double L)
{

    // int block_size = N;
    // int num_blocks = N;

    dim3 block(1);
    dim3 grid(1);


    for(size_t i = 0; i < N-1; i++ ) {
    forces <<<grid, block>>> (rx, ry, rz, fx, fy, fz, epot, pres, temp, rho,
                              V, L, i);

    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());
    }
}

