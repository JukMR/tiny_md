#include "hip/hip_runtime.h"
// Variables necesarias

// #include <hip/hip_runtime.h> // no se porque esto rompe todo

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" // checkCudaError
#include "parameters.h"
#include <cstdio>
// #include <hip/hip_runtime.h>

// #include <hipcub/hipcub.hpp>



//#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

__device__ void minimum_image(float cordi, const float cell_length, float* result)
{
    // imagen más cercana
    if (cordi <= -0.5 * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5 * cell_length) {
        cordi -= cell_length;
    }

    *result = cordi;
}

// Esta función esta sacada de la wiki de Cuda. capaz sirve.


// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
// old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
//     } while (assumed != old);
//     return __longlong_as_double(old);
// }


__device__ double atomicAdd2(double* address, double val) {
unsigned long long int* address_as_ull = (unsigned long long int*)address;
unsigned long long int old = *address_as_ull, assumed;

            do {
                  assumed = old;
                  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val+__longlong_as_double(assumed)));
           } while (assumed != old);
           return __longlong_as_double(old);
}

// Algo asi capaz si se podria implementar para la reduccion con floats.


// __global__ void sum_shared_atomic(const int *in, int n, int *out)
// {
//     __shared__ int partial_sum;

//     uint i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (threadIdx.x == 0) {
//         partial_sum = 0;
//     }

//     __syncthreads();

//     if (i < n) {
//         atomicAdd(&partial_sum, in[i]);
//     }

//     __syncthreads();

//     if (threadIdx.x == 0) {
//         atomicAdd(out, partial_sum);
//     }
// }



__global__ void forces(const float* rx,
                       const float* ry,
                       const float* rz,
                       float* fx,
                       float* fy,
                       float* fz,
                       float* epot,
                       float* pres,
                       const float* temp,
                       const float rho,
                       const float V,
                       const float L,
                       const int row)
{

    //        fx[row] = 0.0d;
    //        fy[row] = 0.0d;
    //        fz[row] = 0.0d;

    //    *epot = 0.0;



    // if (threadIdx.x == 0 ){
    // printf("Soy el hilo %i", threadIdx.x); // esto funciona, solo el hilo 0 ejecuta esto
    float rcut2 = RCUT * RCUT;
    const float RCUT12 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;

    const float RCUT6 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;
    const float ECUT = 4.0 * (1 / (RCUT12)-1 / (RCUT6));
    // //#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

    float fxi = 0.0;
    float fyi = 0.0;
    float fzi = 0.0;
    float epot_partial = 0.0;
    float pres_vir_partial = 0.0;

	unsigned int j = blockIdx.x*blockDim.x + threadIdx.x;
	// unsigned int j =  threadIdx.x;



    // Solo ejecutar para contra los vecinos de la derecha.
    // De todas formas no anda bien

    // for (int j = 0; j < (N - 1); j++) {
    for (; j < (N - 1); j++) {
        if (j != row) {
            float xi = rx[row];
            float yi = ry[row];
            float zi = rz[row];

            float xj = rx[j];
            float yj = ry[j];
            float zj = rz[j];

            float rxd = xi - xj;
            float ryd = yi - yj;
            float rzd = zi - zj;

            minimum_image(rxd, L, &rxd);
            minimum_image(ryd, L, &ryd);
            minimum_image(rzd, L, &rzd);

            float rij2 = rxd * rxd + ryd * ryd + rzd * rzd;

            if (rij2 <= rcut2) {
                float r2inv = 1.0 / rij2;
                float r6inv = r2inv * r2inv * r2inv;

                float fr = 24.0 * r2inv * r6inv * (2.0 * r6inv - 1.0);

                fxi += fr * rxd;
                fyi += fr * ryd;
                fzi += fr * rzd;


                epot_partial += 4.0 * r6inv * (r6inv - 1.0) - ECUT;
                pres_vir_partial += fr * rij2;
            }
        }
    }




    // La implementacion de atomicAdd2 mas arriba parece funcionar pero los resultados siguen siendo incorrectos. No va por acá el error?

    atomicAdd(&fx[row], fxi);
    atomicAdd(&fy[row], fyi);
    atomicAdd(&fz[row], fzi);

    atomicAdd(epot, epot_partial / 2);
    atomicAdd(pres, pres_vir_partial / 2 / (V * 3.0));

    // fx[row] += fxi;
    // fy[row] += fyi;
    // fz[row] += fzi;
    // *epot += epot_partial / 2;
    // *pres += pres_vir_partial / 2 / (V * 3.0);


}


int div_ceil(int a, int b) {
    return (a + b - 1) / b;
}

void launch_forces(const float* rx, const float* ry, const float* rz,
                   float* fx, float* fy, float* fz, float* epot,
                   float* pres, const float* temp, const float rho,
                   const float V, const float L)
{

    // Todavía no entiendo que número de bloques y grilla nos conviene usar para el problema


    // Por ahora tomo N-1 hilos para tener un hilo por cada elemento de N
    dim3 block(N-1);

    // Por ahora la misma selección de grilla usando los ejemplos de Charly
    dim3 grid(div_ceil(N-1, block.x));


    // Este for probablemente no tendria que ir, deberiamos lanzar un kernel que haga esto según el hilo en el que esta parado

    float *epot_tmp;
    float *pres_tmp;

    checkCudaError(hipMallocManaged(&epot_tmp, sizeof( float *)));
    checkCudaError(hipMallocManaged(&pres_tmp, sizeof( float *)));

    for(size_t i = 0; i < N-1; i++ ) {

    *epot_tmp = *epot;
    *pres_tmp = *pres;

    forces <<<grid, block>>> (rx, ry, rz, fx, fy, fz, epot_tmp, pres_tmp, temp, rho, V, L, i);

    *epot = *epot_tmp;
    *pres = *pres_tmp;

    }
    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());

    checkCudaError(hipFree(epot_tmp));
    checkCudaError(hipFree(pres_tmp));
}

