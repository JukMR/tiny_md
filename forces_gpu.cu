#include "hip/hip_runtime.h"
// Variables necesarias

// #include <hip/hip_runtime.h> // no se porque esto rompe todo

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" // checkCudaError
#include "parameters.h"
#include <cstdio>
// #include <hip/hip_runtime.h>

// #include <hipcub/hipcub.hpp>



//#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

__device__ void minimum_image(float cordi, const float cell_length, float* result)
{
    // imagen más cercana
    if (cordi <= -0.5 * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5 * cell_length) {
        cordi -= cell_length;
    }

    *result = cordi;
}

// Esta función esta sacada de la wiki de Cuda. capaz sirve.


// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
// old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
//     } while (assumed != old);
//     return __longlong_as_double(old);
// }


__device__ double atomicAdd2(double* address, double val) {
unsigned long long int* address_as_ull = (unsigned long long int*)address;
unsigned long long int old = *address_as_ull, assumed;

            do {
                  assumed = old;
                  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val+__longlong_as_double(assumed)));
           } while (assumed != old);
           return __longlong_as_double(old);
}

// Algo asi capaz si se podria implementar para la reduccion con floats.


// __global__ void sum_shared_atomic(const int *in, int n, int *out)
// {
//     __shared__ int partial_sum;

//     uint i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (threadIdx.x == 0) {
//         partial_sum = 0;
//     }

//     __syncthreads();

//     if (i < n) {
//         atomicAdd(&partial_sum, in[i]);
//     }

//     __syncthreads();

//     if (threadIdx.x == 0) {
//         atomicAdd(out, partial_sum);
//     }
// }



__global__ void forces(const float* rx,
                       const float* ry,
                       const float* rz,
                       float* fx,
                       float* fy,
                       float* fz,
                       float* epot,
                       float* pres,
                       const float* temp,
                       const float rho,
                       const float V,
                       const float L)
{

    for (uint row = 0; row < N-1; row++){

    float rcut2 = RCUT * RCUT;
    const float RCUT12 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;

    const float RCUT6 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;
    const float ECUT = 4.0 * (1 / (RCUT12)-1 / (RCUT6));
    // //#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

    float fxi = 0.0;
    float fyi = 0.0;
    float fzi = 0.0;
    float epot_partial = 0.0;
    float pres_vir_partial = 0.0;

	unsigned int j = blockIdx.x*blockDim.x + threadIdx.x;

        if (j != row) {
            float xi = rx[row];
            float yi = ry[row];
            float zi = rz[row];

            float xj = rx[j];
            float yj = ry[j];
            float zj = rz[j];

            float rxd = xi - xj;
            float ryd = yi - yj;
            float rzd = zi - zj;

            minimum_image(rxd, L, &rxd);
            minimum_image(ryd, L, &ryd);
            minimum_image(rzd, L, &rzd);

            float rij2 = rxd * rxd + ryd * ryd + rzd * rzd;

            if (rij2 <= rcut2) {
                float r2inv = 1.0 / rij2;
                float r6inv = r2inv * r2inv * r2inv;

                float fr = 24.0 * r2inv * r6inv * (2.0 * r6inv - 1.0);

                fxi += fr * rxd;
                fyi += fr * ryd;
                fzi += fr * rzd;


                epot_partial += 4.0 * r6inv * (r6inv - 1.0) - ECUT;
                pres_vir_partial += fr * rij2;
            }
        }


    atomicAdd(&fx[row], fxi);
    atomicAdd(&fy[row], fyi);
    atomicAdd(&fz[row], fzi);

    atomicAdd(epot, epot_partial / 2);
    atomicAdd(pres, pres_vir_partial / 2 / (V * 3.0));

    // fx[row] += fxi;
    // fy[row] += fyi;
    // fz[row] += fzi;
    // *epot += epot_partial / 2;
    // *pres += pres_vir_partial / 2 / (V * 3.0);


    }
}
int div_ceil(int a, int b) {
    return (a + b - 1) / b;
}

void launch_forces(const float* rx, const float* ry, const float* rz,
                   float* fx, float* fy, float* fz, float* epot,
                   float* pres, const float* temp, const float rho,
                   const float V, const float L)
{

    // Todavía no entiendo que número de bloques y grilla nos conviene usar para el problema


    // Por ahora tomo N-1 hilos para tener un hilo por cada elemento de N
    dim3 block(N-1);

    // Por ahora la misma selección de grilla usando los ejemplos de Charly
    dim3 grid(div_ceil(N-1, block.x));


    // Este for probablemente no tendria que ir, deberiamos lanzar un kernel que haga esto según el hilo en el que esta parado



    forces <<<grid, block>>> (rx, ry, rz, fx, fy, fz, epot, pres, temp, rho, V, L);


    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());

}

