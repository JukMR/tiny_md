#include "hip/hip_runtime.h"
// Variables necesarias

// #include <hip/hip_runtime.h> // no se porque esto rompe todo

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" // checkCudaError
#include "parameters.h"
#include <cstdio>
// #include <hip/hip_runtime.h>

// #include <hipcub/hipcub.hpp>



//#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

__device__ void minimum_image(double cordi, const double cell_length, double* result)
{
    // imagen más cercana
    if (cordi <= -0.5 * cell_length) {
        cordi += cell_length;
    } else if (cordi > 0.5 * cell_length) {
        cordi -= cell_length;
    }

    *result = cordi;
}

// Esta función esta sacada de la wiki de Cuda. capaz sirve.


// __device__ double atomicAdd(double* address, double val)
// {
//     unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
// old = atomicCAS(address_as_ull, assumed,
//                         __double_as_longlong(val +
//                                __longlong_as_double(assumed)));
//     } while (assumed != old);
//     return __longlong_as_double(old);
// }


__device__ double atomicAdd2(double* address, double val) {
unsigned long long int* address_as_ull = (unsigned long long int*)address;
unsigned long long int old = *address_as_ull, assumed;

            do {
                  assumed = old;
                  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val+__longlong_as_double(assumed)));
           } while (assumed != old);
           return __longlong_as_double(old);
}

// Algo asi capaz si se podria implementar para la reduccion con doubles.


// __global__ void sum_shared_atomic(const int *in, int n, int *out)
// {
//     __shared__ int partial_sum;

//     uint i = blockIdx.x * blockDim.x + threadIdx.x;

//     if (threadIdx.x == 0) {
//         partial_sum = 0;
//     }

//     __syncthreads();

//     if (i < n) {
//         atomicAdd(&partial_sum, in[i]);
//     }

//     __syncthreads();

//     if (threadIdx.x == 0) {
//         atomicAdd(out, partial_sum);
//     }
// }



__global__ void forces(const double* rx,
                       const double* ry,
                       const double* rz,
                       double* fx,
                       double* fy,
                       double* fz,
                       double* epot,
                       double* pres,
                       const double* temp,
                       const double rho,
                       const double V,
                       const double L
                       )
{

    //        fx[row] = 0.0d;
    //        fy[row] = 0.0d;
    //        fz[row] = 0.0d;

    //    *epot = 0.0;



    double rcut2 = RCUT * RCUT;
    const double RCUT12 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;

    const double RCUT6 = RCUT * RCUT * RCUT * RCUT * RCUT * RCUT;
    const double ECUT = 4.0 * (1 / (RCUT12)-1 / (RCUT6));
    // //#define ECUT (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)))

    double fxi = 0.0;
    double fyi = 0.0;
    double fzi = 0.0;
    double epot_partial = 0.0;
    double pres_vir_partial = 0.0;


    unsigned int j =  threadIdx.x;
    unsigned int row =  blockIdx.x;


        if (j != row) {
            double xi = rx[row];
            double yi = ry[row];
            double zi = rz[row];

            double xj = rx[j];
            double yj = ry[j];
            double zj = rz[j];

            double rxd = xi - xj;
            double ryd = yi - yj;
            double rzd = zi - zj;

            minimum_image(rxd, L, &rxd);
            minimum_image(ryd, L, &ryd);
            minimum_image(rzd, L, &rzd);

            double rij2 = rxd * rxd + ryd * ryd + rzd * rzd;

            if (rij2 <= rcut2) {
                double r2inv = 1.0 / rij2;
                double r6inv = r2inv * r2inv * r2inv;

                double fr = 24.0 * r2inv * r6inv * (2.0 * r6inv - 1.0);

                fxi += fr * rxd;
                fyi += fr * ryd;
                fzi += fr * rzd;


                epot_partial += 4.0 * r6inv * (r6inv - 1.0) - ECUT;
                pres_vir_partial += fr * rij2;
            // }
        }
    }




    // La implementacion de atomicAdd2 mas arriba parece funcionar pero los resultados siguen siendo incorrectos. No va por acá el error?

    atomicAdd(&fx[row], fxi);
    atomicAdd(&fy[row], fyi);
    atomicAdd(&fz[row], fzi);

    atomicAdd(epot, epot_partial / 2);
    atomicAdd(pres, pres_vir_partial / 2 / (V * 3.0));

    // fx[row] += fxi;
    // fy[row] += fyi;
    // fz[row] += fzi;
    // *epot += epot_partial / 2;
    // *pres += pres_vir_partial / 2 / (V * 3.0);


// }

}
int div_ceil(int a, int b) {
    return (a + b - 1) / b;
}

void launch_forces(const double* rx, const double* ry, const double* rz,
                   double* fx, double* fy, double* fz, double* epot,
                   double* pres, const double* temp, const double rho,
                   const double V, const double L)
{

    // Todavía no entiendo que número de bloques y grilla nos conviene usar para el problema


    // Por ahora tomo N-1 hilos para tener un hilo por cada elemento de N
    dim3 block(N-1);

    // Por ahora la misma selección de grilla usando los ejemplos de Charly
    dim3 grid(N-1);


    // Este for probablemente no tendria que ir, deberiamos lanzar un kernel que haga esto según el hilo en el que esta parado
    // for(size_t i = 0; i < N-1; i++ ) {
    forces <<<grid, block>>> (rx, ry, rz, fx, fy, fz, epot, pres, temp, rho,
                              V, L);

    // }
    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
}

