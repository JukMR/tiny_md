#include "hip/hip_runtime.h"
#ifndef _XOPEN_SOURCE
#define _XOPEN_SOURCE 500 // M_PI
#endif
#include "core.h"
#include "parameters.h"
#include "wtime.h"

#include <cmath>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include "forces_gpu.h"
#include "hip/hip_runtime_api.h"

int main()
{
    FILE *file_xyz, *file_thermo;
    file_xyz = fopen("trajectory.xyz", "w");
    file_thermo = fopen("thermo.log", "w");
    double Ekin, Epot, Temp, Pres; // variables macroscopicas
    double Rho, cell_V, cell_L, tail, Etail, Ptail;
    double *rx, *ry, *rz, *vx, *vy, *vz, *fx, *fy, *fz; // variables microscopicas

    checkCudaError(hipMallocManaged(&rx, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&ry, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&rz, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&vx, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&vy, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&vz, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&fx, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&fy, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&fz, N * sizeof(double *)));


    checkCudaError(hipMemset(rx, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(ry, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(rz, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(vx, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(vy, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(vz, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(fx, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(fy, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(fz, 0, N * sizeof(double *)));

    // rx = (double*)malloc(N * sizeof(double));
    // ry = (double*)malloc(N * sizeof(double));
    // rz = (double*)malloc(N * sizeof(double));
    // vx = (double*)malloc(N * sizeof(double));
    // vy = (double*)malloc(N * sizeof(double));
    // vz = (double*)malloc(N * sizeof(double));
    // fx = (double*)malloc(N * sizeof(double));
    // fy = (double*)malloc(N * sizeof(double));
    // fz = (double*)malloc(N * sizeof(double));

    //    rxyz = (double*)malloc(3 * N * sizeof(double));
    //    vxyz = (double*)malloc(3 * N * sizeof(double));
    //    fxyz = (double*)malloc(3 * N * sizeof(double));

    printf("# Número de partículas:      %d\n", N);
    printf("# Temperatura de referencia: %.2f\n", T0);
    printf("# Pasos de equilibración:    %d\n", TEQ);
    printf("# Pasos de medición:         %d\n", TRUN - TEQ);
    printf("# (mediciones cada %d pasos)\n", TMES);
    printf("# densidad, volumen, energía potencial media, presión media\n");
    fprintf(file_thermo, "# t Temp Pres Epot Etot\n");

    srand(SEED);
    double t = 0.0, sf;
    double Rhob;
    Rho = RHOI;
    init_pos(rx, ry, rz, Rho);
    double start = wtime();

    // double ecut = (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)));
    for (int m = 0; m < 9; m++) {
        Rhob = Rho;
        Rho = RHOI - 0.1 * (double)m;
        cell_V = (double)N / Rho;
        cell_L = cbrt(cell_V);
        tail = 16.0 * M_PI * Rho * ((2.0 / 3.0) * pow(RCUT, -9) - pow(RCUT, -3)) / 3.0;
        Etail = tail * (double)N;
        Ptail = tail * Rho;

        int i = 0;
        sf = cbrt(Rhob / Rho);

            for (int k = 0; k < N; k++) { // reescaleo posiciones a nueva densidad
                rx[k] *= sf;
                ry[k] *= sf;
                rz[k] *= sf;
            }

            init_vel(vx, vy, vz, &Temp, &Ekin);

	    for (int j = 0; j <  N; j++) {
              fx[j] = 0.0;
              fy[j] = 0.0;
              fz[j] = 0.0;
            }
            Epot=0;
            Pres=Temp* Rho ;


            double *epot_aux;
            double *pres_aux;
            double *ptr_Temp;


            checkCudaError(hipMallocManaged(&epot_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&pres_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&ptr_Temp, sizeof(double *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = Temp;

            // for (int i = 0; i < N-1; i+=1){
                launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, Rho, cell_V, cell_L); // actualizo fuerzas

                Temp = *ptr_Temp;
            // }

             Epot+=*epot_aux;
             Pres+=*pres_aux;

            checkCudaError(hipFree(epot_aux));
            checkCudaError(hipFree(pres_aux));
            checkCudaError(hipFree(ptr_Temp));


        for (i = 1; i < TEQ; i++) { // loop de equilibracion

            velocity_verlet(rx, ry, rz, vx, vy, vz, fx, fy, fz, &Epot, &Ekin, &Pres, &Temp, Rho, cell_V, cell_L);

            sf = sqrt(T0 / Temp);
            for (int k = 0; k < N; k++) { // reescaleo de velocidades
                vx[k] *= sf;
                vy[k] *= sf;
                vz[k] *= sf;
            }
        }

        int mes = 0;
        double epotm = 0.0, presm = 0.0;
        for (i = TEQ; i < TRUN; i++) { // loop de medicion

            velocity_verlet(rx, ry, rz, vx, vy, vz, fx, fy, fz, &Epot, &Ekin, &Pres, &Temp, Rho, cell_V, cell_L);

            sf = sqrt(T0 / Temp);
            for (int k = 0; k < N; k++) { // reescaleo de velocidades
                vx[k] *= sf;
                vy[k] *= sf;
                vz[k] *= sf;
            }

            if (i % TMES == 0) {
                Epot += Etail;
                Pres += Ptail;

                epotm += Epot;
                presm += Pres;
                mes++;

                fprintf(file_thermo, "%f %f %f %f %f\n", t, Temp, Pres, Epot, Epot + Ekin);
                fprintf(file_xyz, "%d\n\n", N);
                for (int k = 0; k < N; k++) {
                    fprintf(file_xyz, "Ar %e %e %e\n", rx[k], ry[k], rz[k]);
                }
            }

            t += DT;
        }
        printf("%f\t%f\t%f\t%f\n", Rho, cell_V, epotm / (double)mes, presm / (double)mes);
    }

    double elapsed = wtime() - start;
    FILE* logs;
    logs = fopen("statics.res", "a");
    if (logs == NULL) {
        printf("Cannot open statics log file");
        exit(EXIT_FAILURE);
    }

    fprintf(logs, "# Tiempo total de simulación = %f segundos\n", elapsed);
    double foperations = (N * (N - 1) * 0.5 * 41.0 + 5.0) * TRUN;
    fprintf(logs, "%s %f \n", "Floating point operation done:", foperations);
    double flops = foperations / elapsed;
    fprintf(logs, "%s %f\n", "FLOPS:", flops);
    fprintf(logs, "%s %f\n", "GFLOPS:", flops / (1000.0 * 1000.0 * 1000.0));
    fprintf(logs, "# Tiempo simulado = %f [fs]\n", t * 1.6);
    fprintf(logs, "# ns/day = %f\n", (1.6e-6 * t) / elapsed * 86400);
    //                       ^1.6 fs -> ns       ^sec -> day


    checkCudaError(hipFree(rx));
    checkCudaError(hipFree(ry));
    checkCudaError(hipFree(rz));
    checkCudaError(hipFree(vx));
    checkCudaError(hipFree(vy));
    checkCudaError(hipFree(vz));
    checkCudaError(hipFree(fx));
    checkCudaError(hipFree(fy));
    checkCudaError(hipFree(fz));

    return 0;
}
