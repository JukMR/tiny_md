#include "hip/hip_runtime.h"
#ifndef _XOPEN_SOURCE
#define _XOPEN_SOURCE 500 // M_PI
#endif
#include "core.h"
#include "parameters.h"
#include "wtime.h"

#include <cmath>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include "forces_gpu.h"
#include "hip/hip_runtime_api.h"

int main()
{
    FILE *file_xyz, *file_thermo;
    file_xyz = fopen("trajectory.xyz", "w");
    file_thermo = fopen("thermo.log", "w");
    float Ekin, Epot, Temp, Pres; // variables macroscopicas
    float Rho, cell_V, cell_L, tail, Etail, Ptail;
    float *rx, *ry, *rz, *vx, *vy, *vz, *fx, *fy, *fz; // variables microscopicas

    checkCudaError(hipMallocManaged(&rx, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&ry, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&rz, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&vx, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&vy, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&vz, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&fx, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&fy, N * sizeof(float *)));
    checkCudaError(hipMallocManaged(&fz, N * sizeof(float *)));


    checkCudaError(hipMemset(rx, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(ry, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(rz, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(vx, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(vy, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(vz, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(fx, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(fy, 0, N * sizeof(float *)));
    checkCudaError(hipMemset(fz, 0, N * sizeof(float *)));

    // rx = (float*)malloc(N * sizeof(float));
    // ry = (float*)malloc(N * sizeof(float));
    // rz = (float*)malloc(N * sizeof(float));
    // vx = (float*)malloc(N * sizeof(float));
    // vy = (float*)malloc(N * sizeof(float));
    // vz = (float*)malloc(N * sizeof(float));
    // fx = (float*)malloc(N * sizeof(float));
    // fy = (float*)malloc(N * sizeof(float));
    // fz = (float*)malloc(N * sizeof(float));

    //    rxyz = (float*)malloc(3 * N * sizeof(float));
    //    vxyz = (float*)malloc(3 * N * sizeof(float));
    //    fxyz = (float*)malloc(3 * N * sizeof(float));

    printf("# Número de partículas:      %d\n", N);
    printf("# Temperatura de referencia: %.2f\n", T0);
    printf("# Pasos de equilibración:    %d\n", TEQ);
    printf("# Pasos de medición:         %d\n", TRUN - TEQ);
    printf("# (mediciones cada %d pasos)\n", TMES);
    printf("# densidad, volumen, energía potencial media, presión media\n");
    fprintf(file_thermo, "# t Temp Pres Epot Etot\n");

    srand(SEED);
    float t = 0.0, sf;
    float Rhob;
    Rho = RHOI;
    init_pos(rx, ry, rz, Rho);
    float start = wtime();

    // float ecut = (4.0 * (pow(RCUT, -12) - pow(RCUT, -6)));
    for (int m = 0; m < 9; m++) {
        Rhob = Rho;
        Rho = RHOI - 0.1 * (float)m;
        cell_V = (float)N / Rho;
        cell_L = cbrt(cell_V);
        tail = 16.0 * M_PI * Rho * ((2.0 / 3.0) * pow(RCUT, -9) - pow(RCUT, -3)) / 3.0;
        Etail = tail * (float)N;
        Ptail = tail * Rho;

        int i = 0;
        sf = cbrt(Rhob / Rho);

            for (int k = 0; k < N; k++) { // reescaleo posiciones a nueva densidad
                rx[k] *= sf;
                ry[k] *= sf;
                rz[k] *= sf;
            }

            init_vel(vx, vy, vz, &Temp, &Ekin);

	    for (int j = 0; j <  N; j++) {
              fx[j] = 0.0;
              fy[j] = 0.0;
              fz[j] = 0.0;
            }
            Epot=0;
            Pres=Temp* Rho ;


            float *epot_aux;
            float *pres_aux;
            float *ptr_Temp;


            checkCudaError(hipMallocManaged(&epot_aux, sizeof(float *)));
            checkCudaError(hipMallocManaged(&pres_aux, sizeof(float *)));
            checkCudaError(hipMallocManaged(&ptr_Temp, sizeof(float *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = Temp;

            // for (int i = 0; i < N-1; i+=1){
                launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, Rho, cell_V, cell_L); // actualizo fuerzas

                Temp = *ptr_Temp;
            // }

             Epot+=*epot_aux;
             Pres+=*pres_aux;

            checkCudaError(hipFree(epot_aux));
            checkCudaError(hipFree(pres_aux));
            checkCudaError(hipFree(ptr_Temp));


        for (i = 1; i < TEQ; i++) { // loop de equilibracion

            velocity_verlet(rx, ry, rz, vx, vy, vz, fx, fy, fz, &Epot, &Ekin, &Pres, &Temp, Rho, cell_V, cell_L);

            sf = sqrt(T0 / Temp);
            for (int k = 0; k < N; k++) { // reescaleo de velocidades
                vx[k] *= sf;
                vy[k] *= sf;
                vz[k] *= sf;
            }
        }

        int mes = 0;
        float epotm = 0.0, presm = 0.0;
        for (i = TEQ; i < TRUN; i++) { // loop de medicion

            velocity_verlet(rx, ry, rz, vx, vy, vz, fx, fy, fz, &Epot, &Ekin, &Pres, &Temp, Rho, cell_V, cell_L);

            sf = sqrt(T0 / Temp);
            for (int k = 0; k < N; k++) { // reescaleo de velocidades
                vx[k] *= sf;
                vy[k] *= sf;
                vz[k] *= sf;
            }

            if (i % TMES == 0) {
                Epot += Etail;
                Pres += Ptail;

                epotm += Epot;
                presm += Pres;
                mes++;

                fprintf(file_thermo, "%f %f %f %f %f\n", t, Temp, Pres, Epot, Epot + Ekin);
                fprintf(file_xyz, "%d\n\n", N);
                for (int k = 0; k < N; k++) {
                    fprintf(file_xyz, "Ar %e %e %e\n", rx[k], ry[k], rz[k]);
                }
            }

            t += DT;
        }
        printf("%f\t%f\t%f\t%f\n", Rho, cell_V, epotm / (float)mes, presm / (float)mes);
    }

    float elapsed = wtime() - start;
    FILE* logs;
    logs = fopen("statics.res", "a");
    if (logs == NULL) {
        printf("Cannot open statics log file");
        exit(EXIT_FAILURE);
    }

    fprintf(logs, "# Tiempo total de simulación = %f segundos\n", elapsed);
    float foperations = (N * (N - 1) * 0.5 * 41.0 + 5.0) * TRUN;
    fprintf(logs, "%s %f \n", "Floating point operation done:", foperations);
    float flops = foperations / elapsed;
    fprintf(logs, "%s %f\n", "FLOPS:", flops);
    fprintf(logs, "%s %f\n", "GFLOPS:", flops / (1000.0 * 1000.0 * 1000.0));
    fprintf(logs, "# Tiempo simulado = %f [fs]\n", t * 1.6);
    fprintf(logs, "# ns/day = %f\n", (1.6e-6 * t) / elapsed * 86400);
    //                       ^1.6 fs -> ns       ^sec -> day


    checkCudaError(hipFree(rx));
    checkCudaError(hipFree(ry));
    checkCudaError(hipFree(rz));
    checkCudaError(hipFree(vx));
    checkCudaError(hipFree(vy));
    checkCudaError(hipFree(vz));
    checkCudaError(hipFree(fx));
    checkCudaError(hipFree(fy));
    checkCudaError(hipFree(fz));

    return 0;
}
