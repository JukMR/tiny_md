#include "hip/hip_runtime.h"
#ifndef _XOPEN_SOURCE
#define _XOPEN_SOURCE 500 // M_PI
#endif

#include "core.h"
#include "parameters.h"

#include <GL/glut.h> // OpenGL
#include <math.h>
#include <stdio.h>
#include <stdlib.h>


#include <hip/hip_runtime.h>
#include "forces_gpu.h"
#include "hip/hip_runtime_api.h"

// variables globales
static double Ekin, Epot, Temp, Pres; // variables macroscopicas
static double Rho, V, box_size, tail, Etail, Ptail;
static double *rx, *ry, *rz, *vx, *vy, *vz, *fx, *fy, *fz; // variables microscopicas
static double Rhob, sf, epotm, presm;
static int switcher = 0, frames = 0, mes;


// OpenGL specific drawing routines
static int win_id;
static int win_x = 900, win_y = 900;


static void pre_display(void)
{ // 3D
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    gluPerspective(45.0, (float)win_x / win_y, 1.0, 0.0);
    gluLookAt(1.0, 1.0, 1.0, 0.5, 0.5, 0.5, 1.0, 0.0, 0.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
}


static void post_display(void)
{
    glutSwapBuffers();
}


static void draw_atoms(void)
{
    double glL = cbrt((double)N / (RHOI - 0.8));

    double resize = 0.5;

    // grafico las lineas que delimitan la caja de simulación
    glBegin(GL_LINES);

    double box_line = resize * (box_size / glL);
    glColor3d(0.0, 0.0, 1.0);

    glVertex3d(0.0, 0.0, 0.0);
    glVertex3d(0.0, 0.0, box_line);

    glVertex3d(0.0, 0.0, 0.0);
    glVertex3d(0.0, box_line, 0.0);

    glVertex3d(0.0, 0.0, 0.0);
    glVertex3d(box_line, 0.0, 0.0);

    glVertex3d(box_line, box_line, box_line);
    glVertex3d(box_line, box_line, 0.0);

    glVertex3d(box_line, box_line, box_line);
    glVertex3d(box_line, 0.0, box_line);

    glVertex3d(box_line, box_line, box_line);
    glVertex3d(0.0, box_line, box_line);

    glVertex3d(0.0, box_line, 0.0);
    glVertex3d(box_line, box_line, 0.0);

    glVertex3d(0.0, box_line, box_line);
    glVertex3d(0.0, 0.0, box_line);

    glVertex3d(box_line, 0.0, box_line);
    glVertex3d(box_line, 0.0, 0.0);

    glVertex3d(box_line, 0.0, box_line);
    glVertex3d(0.0, 0.0, box_line);

    glVertex3d(0.0, box_line, box_line);
    glVertex3d(0.0, box_line, 0.0);

    glVertex3d(box_line, box_line, 0.0);
    glVertex3d(box_line, 0.0, 0.0);

    glEnd();

    // grafico las particulas (x, y, z) en el punto (dx, dy, dx), son reescaleadas
    // a [0, 1] y luego multiplicadas con un factor que las achica para poder
    // apreciar mejor el cambio en el volumen
    glBegin(GL_POINTS);

    int di;

    double dx;
    double dy;
    double dz;

    for (di = 0; di < N; di++) {
        dx = (rx[di] / glL) * resize;
        dy = (ry[di] / glL) * resize;
        dz = (rz[di] / glL) * resize;

        glColor3d(0.0, 1.0, 0.0);
        glVertex3d(dx, dy, dz);
    }

    glEnd();
}


/*static void reshape_func ( int width, int height )
{
	glutSetWindow ( win_id );
	glutReshapeWindow ( width, height );

	win_x = width;
	win_y = height;
}*/


static void idle_func(void)
{

    if (switcher == 3) {

        Rho = RHOI;
        V = (double)N / Rho;
        box_size = cbrt(V);
        tail = 16.0 * M_PI * Rho * ((2.0 / 3.0) * pow(RCUT, -9) - pow(RCUT, -3)) / 3.0;
        Etail = tail * (double)N;
        Ptail = tail * Rho;

        init_pos(rx, ry, rz, Rho);
        init_vel(vx, vy, vz, &Temp, &Ekin);

        for (int j = 0; j < N; j++) {
            fx[j] = 0.0;
            fy[j] = 0.0;
            fz[j] = 0.0;
        }
        Epot = 0;
        Pres = Temp * Rho;
        // #pragma omp parallel
        {
            double *epot_aux;
            double *pres_aux;
            double *ptr_Temp;


            checkCudaError(hipMallocManaged(&epot_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&pres_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&ptr_Temp, sizeof(double *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = Temp;

            // for (int i = 0; i < N - 1; i += 1) {
                launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, Rho, V, box_size);
            // }
            Epot += *epot_aux;
            Pres += *pres_aux;

            checkCudaError(hipFree(epot_aux));
            checkCudaError(hipFree(pres_aux));
            checkCudaError(hipFree(ptr_Temp));
        }

        switcher = 0;

    } else if (switcher == 2) { // imprimo propiedades en la terminal y cambio la densidad

        printf("%f\t%f\t%f\t%f\n", Rho, V, epotm / (double)mes,
               presm / (double)mes);

        Rhob = Rho;
        Rho = Rho - 0.1;

        V = (double)N / Rho;
        box_size = cbrt(V);
        tail = 16.0 * M_PI * Rho * ((2.0 / 3.0) * pow(RCUT, -9) - pow(RCUT, -3)) / 3.0;
        Etail = tail * (double)N;
        Ptail = tail * Rho;

        sf = cbrt(Rhob / Rho);
        for (int k = 0; k < N; k++) { // reescaleo posiciones a nueva densidad
            rx[k] *= sf;
            ry[k] *= sf;
            rz[k] *= sf;
        }
        init_vel(vx, vy, vz, &Temp, &Ekin);

        for (int j = 0; j < N; j++) {
            fx[j] = 0.0;
            fy[j] = 0.0;
            fz[j] = 0.0;
        }
        Epot = 0;
        Pres = Temp * Rho;
        // #pragma omp parallel
        {
            double *epot_aux;
            double *pres_aux;
            double *ptr_Temp;


            checkCudaError(hipMallocManaged(&epot_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&pres_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&ptr_Temp, sizeof(double *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = Temp;

            // #pragma omp for
            // for (int i = 0; i < N - 1; i += 1) {
                launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, Rho, V, box_size);
            // }
            // #pragma omp critical
            Epot += *epot_aux;
            Pres += *pres_aux;

            checkCudaError(hipFree(epot_aux));
            checkCudaError(hipFree(pres_aux));
            checkCudaError(hipFree(ptr_Temp));
        }

        switcher = 0;
        if (fabs(Rho - (RHOI - 0.9f)) < 1e-6) {
            printf("\n");
            switcher = 3;
        }

    } else if (switcher == 1) { // loop de medición


        for (int i = frames; i < frames + TMES; i++) {

            velocity_verlet(rx, ry, rz, vx, vy, vz, fx, fy, fz, &Epot, &Ekin, &Pres, &Temp, Rho, V, box_size);

            sf = sqrt(T0 / Temp);
            for (int k = 0; k < N; k++) { // reescaleo de velocidades
                vx[k] *= sf;
                vy[k] *= sf;
                vz[k] *= sf;
            }
        }

        Epot += Etail;
        Pres += Ptail;

        epotm += Epot;
        presm += Pres;
        mes++;

        frames += TMES;
        if (frames % TRUN == 0) {
            switcher = 2;
        }

    } else if (switcher == 0) { // loop de equilibración


        while (frames % TEQ != 0) {

            velocity_verlet(rx, ry, rz, vx, vy, vz, fx, fy, fz, &Epot, &Ekin, &Pres, &Temp, Rho, V, box_size);

            sf = sqrt(T0 / Temp);
            for (int k = 0; k < N; k++) { // reescaleo de velocidades
                vx[k] *= sf;
                vy[k] *= sf;
                vz[k] *= sf;
            }

            frames++;
        }

        mes = 0;
        epotm = 0.0;
        presm = 0.0;

        switcher = 1;
    }
    glutSetWindow(win_id);
    glutPostRedisplay();
}


static void display_func(void)
{
    pre_display();
    draw_atoms();
    post_display();
}


static void open_glut_window(void)
{
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);

    glutInitWindowPosition(0, 0);
    glutInitWindowSize(win_x, win_y);
    win_id = glutCreateWindow("tiny molecular dynamics | visualization");

    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    glutSwapBuffers();
    glClear(GL_COLOR_BUFFER_BIT);
    glutSwapBuffers();

    pre_display();

    // glutKeyboardFunc ( key_func );
    // glutMouseFunc ( mouse_func );
    // glutMotionFunc ( motion_func );
    //glutReshapeFunc ( reshape_func );

    glutIdleFunc(idle_func);
    glutDisplayFunc(display_func);
}


// viz main

int main(int argc, char** argv)
{
    glutInit(&argc, argv);

    // rx = (double*)malloc(N * sizeof(double));
    // ry = (double*)malloc(N * sizeof(double));
    // rz = (double*)malloc(N * sizeof(double));
    // vx = (double*)malloc(N * sizeof(double));
    // vy = (double*)malloc(N * sizeof(double));
    // vz = (double*)malloc(N * sizeof(double));
    // fx = (double*)malloc(N * sizeof(double));
    // fy = (double*)malloc(N * sizeof(double));
    // fz = (double*)malloc(N * sizeof(double));

    checkCudaError(hipMallocManaged(&rx, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&ry, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&rz, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&vx, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&vy, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&vz, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&fx, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&fy, N * sizeof(double *)));
    checkCudaError(hipMallocManaged(&fz, N * sizeof(double *)));


    checkCudaError(hipMemset(rx, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(ry, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(rz, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(vx, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(vy, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(vz, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(fx, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(fy, 0, N * sizeof(double *)));
    checkCudaError(hipMemset(fz, 0, N * sizeof(double *)));


    // parametros iniciales para que los pueda usar (antes de modificar)
    // `idle_func`
    srand(SEED);
    Rho = RHOI;
    Rhob = Rho;
    V = (double)N / Rho;
    box_size = cbrt(V);
    tail = 16.0 * M_PI * Rho * ((2.0 / 3.0) * pow(RCUT, -9) - pow(RCUT, -3)) / 3.0;
    Etail = tail * (double)N;
    Ptail = tail * Rho;

    init_pos(rx, ry, rz, Rho);
    init_vel(vx, vy, vz, &Temp, &Ekin);

    for (int j = 0; j < N; j++) {
        fx[j] = 0.0;
        fy[j] = 0.0;
        fz[j] = 0.0;
    }
    Epot = 0;
    Pres = Temp * Rho;
    // #pragma omp parallel
        {
            double *epot_aux;
            double *pres_aux;
            double *ptr_Temp;


            checkCudaError(hipMallocManaged(&epot_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&pres_aux, sizeof(double *)));
            checkCudaError(hipMallocManaged(&ptr_Temp, sizeof(double *)));

            *epot_aux=0;
            *pres_aux=0;
            *ptr_Temp = Temp;

            // #pragma omp for
            // for (int i = 0; i < N - 1; i += 1) {
                launch_forces(rx, ry, rz, fx, fy, fz, epot_aux, pres_aux, ptr_Temp, Rho, V, box_size);
            // }
            // #pragma omp critical
            Epot += *epot_aux;
            Pres += *pres_aux;

            checkCudaError(hipFree(epot_aux));
            checkCudaError(hipFree(pres_aux));
            checkCudaError(hipFree(ptr_Temp));
        }


    printf("# Número de partículas:      %d\n", N);
    printf("# Temperatura de referencia: %.2f\n", T0);
    printf("# Pasos de equilibración:    %d\n", TEQ);
    printf("# Pasos de medición:         %d\n", TRUN - TEQ);
    printf("# (mediciones cada %d pasos)\n", TMES);

    open_glut_window();

    glutMainLoop();

    checkCudaError(hipFree(rx));
    checkCudaError(hipFree(ry));
    checkCudaError(hipFree(rz));
    checkCudaError(hipFree(vx));
    checkCudaError(hipFree(vy));
    checkCudaError(hipFree(vz));
    checkCudaError(hipFree(fx));
    checkCudaError(hipFree(fy));
    checkCudaError(hipFree(fz));

    exit(0);
}
